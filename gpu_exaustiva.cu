#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <iomanip>
#include <tuple>

using namespace std;

struct up_comp
{
    int up, l_up;
    char chr;
    up_comp(int chr_) : chr(chr_){};
    __host__ __device__ int operator()(const thrust::tuple<char, int, int> &tup)
    {
        int act_val = thrust::get<0>(tup);

        if (chr == thrust::get<2>(tup))
        {
            l_up = act_val + 2;
        }
        else
        {
            l_up = act_val - 1;
        }
        up = thrust::get<1>(tup) - 1;
        if (l_up > up && l_up > 0)
        {
            return l_up;
        }
        else if (up >= l_up && up > 0)
        {
            return up;
        }
        return act_val;
    }
};

struct left_comp
{
    __host__ __device__ int operator()(const int &x, const int &y)
    {
        int left, act;
        left = x;
        act = y;

        if (act >= left && act > 0)
        {
            return act;
        }
        else if (left > act && left > 0)
        {
            return left;
        }
        return 0;
    }
};

int main()
{
    int m, n;
    int as, bs, max, fixMax;
    string seq1, seq2;

    cin >> m;
    cin >> n;

    bs = -(m + n);

    cin >> seq1;
    cin >> seq2;

    seq1 = '_' + seq1;
    seq2 = '_' + seq2;

    thrust::device_vector<char> seq1_GPU(m + 1);
    thrust::device_vector<char> seq2_GPU(n + 1);
    thrust::device_vector<int> res1(n + 1);
    thrust::device_vector<int> copy_res1(n + 1);

    if (m > n)
    {
        max = n;
     fixMax = n;
    }
    else
    {
        max = m;
     fixMax = m;
    }

    for (int i = 0; i < m + 1; i++)
    {
        seq1_GPU[i] = seq1[i];
    }
    for (int j = 0; j < n + 1; j++)
    {
        seq2_GPU[j] = seq2[j];
    }

    thrust::fill(copy_res1.begin(), copy_res1.end(), 0);

    while (max > 0)
    {
        for (int i = 0; i <= (m + 1 - max); i++)
        {
            for (int j = 0; j <= (n + 1 - max); j++)
            {
                thrust::fill(res1.begin(), res1.end(), 0);
                for (int k = 0; k <= max; k++)
                {
                    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(res1.begin(),
                                                                                   res1.begin() + 1,
                                                                                  seq2_GPU.begin() + 1 + j)),
                                      thrust::make_zip_iterator(thrust::make_tuple(res1.begin() + max,
                                                                                   res1.begin() + 1 + max,
                                                                                    seq2_GPU.end() + 1 + j + max)),
                                      copy_res1.begin() + 1,
                                      up_comp(seq1_GPU[k]));

                    thrust::inclusive_scan(copy_res1.begin(), copy_res1.begin() + max, res1.begin(), left_comp());

                    as = thrust::reduce(res1.begin(), res1.end(), 0, thrust::maximum<int>());

                    if (as > bs)
                    {
                        bs = as;
                    }
                }
            }
        }
        max -= 1;
    }
    if(seq1[1]==seq2[1]) bs += 2;
    cout << bs << endl;
    return 0;
}